
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2017 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "kernel_config.h"

#include "kernel/kernel_compat_cuda.h"

#include "kernel/filter/filter_kernel.h"

/* kernels */

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_shadow(int sample,
                                 TilesInfo *tiles,
                                 float *unfilteredA,
                                 float *unfilteredB,
                                 float *sampleVariance,
                                 float *sampleVarianceV,
                                 float *bufferVariance,
                                 int4 prefilter_rect,
                                 int buffer_pass_stride,
                                 int buffer_denoising_offset,
                                 bool use_split_variance)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_divide_shadow(sample,
		                            tiles,
		                            x, y,
		                            unfilteredA,
		                            unfilteredB,
		                            sampleVariance,
		                            sampleVarianceV,
		                            bufferVariance,
		                            prefilter_rect,
		                            buffer_pass_stride,
		                            buffer_denoising_offset,
		                            use_split_variance);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_get_feature(int sample,
                               TilesInfo *tiles,
                               int m_offset,
                               int v_offset,
                               float *mean,
                               float *variance,
                               int4 prefilter_rect,
                               int buffer_pass_stride,
                               int buffer_denoising_offset,
                               bool use_split_variance)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_get_feature(sample,
		                          tiles,
		                          m_offset, v_offset,
		                          x, y,
		                          mean, variance,
		                          prefilter_rect,
		                          buffer_pass_stride,
		                          buffer_denoising_offset,
		                          use_split_variance);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_detect_outliers(float *image,
                                   float *variance,
                                   float *depth,
                                   float *output,
                                   int4 prefilter_rect,
                                   int pass_stride)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_detect_outliers(x, y, image, variance, depth, output, prefilter_rect, pass_stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_combine_halves(float *mean, float *variance, float *a, float *b, int4 prefilter_rect, int r)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_combine_halves(x, y, mean, variance, a, b, prefilter_rect, r);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_construct_transform(float const* __restrict__ buffer,
                                       float *transform, int *rank,
                                       int4 filter_area, int4 rect,
                                       int radius, float pca_threshold,
                                       int pass_stride)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int *l_rank = rank + y*filter_area.z + x;
		float *l_transform = transform + y*filter_area.z + x;
		kernel_filter_construct_transform(buffer,
		                                  x + filter_area.x, y + filter_area.y,
		                                  rect, pass_stride,
		                                  l_transform, l_rank,
		                                  radius, pca_threshold,
		                                  filter_area.z*filter_area.w,
		                                  threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_difference(int dx, int dy,
                                       const float *ccl_restrict weight_image,
                                       const float *ccl_restrict variance_image,
                                       float *difference_image,
                                       int4 rect, int w,
                                       int channel_offset,
                                       float a, float k_2)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_calc_difference(x, y, dx, dy, weight_image, variance_image, difference_image, rect, w, channel_offset, a, k_2);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_blur(const float *ccl_restrict difference_image, float *out_image, int4 rect, int w, int f)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_blur(x, y, difference_image, out_image, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_weight(const float *ccl_restrict difference_image, float *out_image, int4 rect, int w, int f)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_calc_weight(x, y, difference_image, out_image, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_update_output(int dx, int dy,
                                     const float *ccl_restrict difference_image,
                                     const float *ccl_restrict image,
                                     float *out_image, float *accum_image,
                                     int4 rect, int w,
                                     int f)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_update_output(x, y, dx, dy, difference_image, image, out_image, accum_image, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_normalize(float *out_image, const float *ccl_restrict accum_image, int4 rect, int w)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_normalize(x, y, out_image, accum_image, rect, w);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_construct_gramian(int dx, int dy,
                                         const float *ccl_restrict difference_image,
                                         const float *ccl_restrict buffer,
                                         float const* __restrict__ transform,
                                         int *rank,
                                         float *XtWX,
                                         float3 *XtWY,
                                         int4 rect,
                                         int4 filter_rect,
                                         int w, int h, int f,
                                         int pass_stride)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x + max(0, rect.x-filter_rect.x);
	int y = blockDim.y*blockIdx.y + threadIdx.y + max(0, rect.y-filter_rect.y);
	if(x < min(filter_rect.z, rect.z-filter_rect.x) && y < min(filter_rect.w, rect.w-filter_rect.y)) {
		kernel_filter_nlm_construct_gramian(x, y,
		                                    dx, dy,
		                                    difference_image,
		                                    buffer,
		                                    transform, rank,
		                                    XtWX, XtWY,
		                                    rect, filter_rect,
		                                    w, h, f,
		                                    pass_stride,
		                                    threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_finalize(int w, int h,
                            float *buffer, int *rank,
                            float *XtWX, float3 *XtWY,
                            int4 filter_area, int4 buffer_params,
                            int sample)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int storage_ofs = y*filter_area.z+x;
		rank += storage_ofs;
		XtWX += storage_ofs;
		XtWY += storage_ofs;
		kernel_filter_finalize(x, y, w, h, buffer, rank, filter_area.z*filter_area.w, XtWX, XtWY, buffer_params, sample);
	}
}

#endif

