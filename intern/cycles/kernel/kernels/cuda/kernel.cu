
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "kernel/kernel_compat_cuda.h"
#include "kernel_config.h"
#include "kernel/kernel_math.h"
#include "kernel/kernel_types.h"
#include "kernel/kernel_globals.h"
#include "kernel/kernel_film.h"
#include "kernel/kernel_path.h"
#include "kernel/kernel_path_branched.h"
#include "kernel/kernel_bake.h"

/* kernels */
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		KernelGlobals kg;
		kernel_path_trace(&kg, buffer, rng_state, sample, x, y, offset, stride);
	}
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_BRANCHED_MAX_REGISTERS)
kernel_cuda_branched_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		KernelGlobals kg;
		kernel_branched_path_trace(&kg, buffer, rng_state, sample, x, y, offset, stride);
	}
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_shader(uint4 *input,
                   float4 *output,
                   float *output_luma,
                   int type,
                   int sx,
                   int sw,
                   int offset,
                   int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_shader_evaluate(&kg,
		                       input,
		                       output,
		                       output_luma,
		                       (ShaderEvalType)type, 
		                       x,
		                       sample);
	}
}

#ifdef __BAKING__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_bake(uint4 *input, float4 *output, int type, int filter, int sx, int sw, int offset, int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_bake_evaluate(&kg, input, output, (ShaderEvalType)type, filter, x, offset, sample);
	}
}
#endif

#endif

