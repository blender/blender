
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2016 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA split kernel entry points */

#ifdef __CUDA_ARCH__

#define __SPLIT_KERNEL__

#include "kernel/kernel_compat_cuda.h"
#include "kernel_config.h"

#include "kernel/split/kernel_split_common.h"
#include "kernel/split/kernel_data_init.h"
#include "kernel/split/kernel_path_init.h"
#include "kernel/split/kernel_scene_intersect.h"
#include "kernel/split/kernel_lamp_emission.h"
#include "kernel/split/kernel_do_volume.h"
#include "kernel/split/kernel_queue_enqueue.h"
#include "kernel/split/kernel_indirect_background.h"
#include "kernel/split/kernel_shader_setup.h"
#include "kernel/split/kernel_shader_sort.h"
#include "kernel/split/kernel_shader_eval.h"
#include "kernel/split/kernel_holdout_emission_blurring_pathtermination_ao.h"
#include "kernel/split/kernel_subsurface_scatter.h"
#include "kernel/split/kernel_direct_lighting.h"
#include "kernel/split/kernel_shadow_blocked_ao.h"
#include "kernel/split/kernel_shadow_blocked_dl.h"
#include "kernel/split/kernel_enqueue_inactive.h"
#include "kernel/split/kernel_next_iteration_setup.h"
#include "kernel/split/kernel_indirect_subsurface.h"
#include "kernel/split/kernel_buffer_update.h"

#include "kernel/kernel_film.h"

/* kernels */
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_state_buffer_size(uint num_threads, uint64_t *size)
{
	*size = split_data_buffer_size(NULL, num_threads);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace_data_init(
        ccl_global void *split_data_buffer,
        int num_elements,
        ccl_global char *ray_state,
        ccl_global uint *rng_state,
        int start_sample,
        int end_sample,
        int sx, int sy, int sw, int sh, int offset, int stride,
        ccl_global int *Queue_index,
        int queuesize,
        ccl_global char *use_queues_flag,
        ccl_global unsigned int *work_pool_wgs,
        unsigned int num_samples,
        ccl_global float *buffer)
{
	kernel_data_init(NULL,
	                 NULL,
	                 split_data_buffer,
	                 num_elements,
	                 ray_state,
	                 rng_state,
	                 start_sample,
	                 end_sample,
	                 sx, sy, sw, sh, offset, stride,
	                 Queue_index,
	                 queuesize,
	                 use_queues_flag,
	                 work_pool_wgs,
	                 num_samples,
	                 buffer);
}

#define DEFINE_SPLIT_KERNEL_FUNCTION(name) \
	extern "C" __global__ void \
	CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS) \
	kernel_cuda_##name() \
	{ \
		kernel_##name(NULL); \
	}

#define DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(name, type) \
	extern "C" __global__ void \
	CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS) \
	kernel_cuda_##name() \
	{ \
		ccl_local type locals; \
		kernel_##name(NULL, &locals); \
	}

DEFINE_SPLIT_KERNEL_FUNCTION(path_init)
DEFINE_SPLIT_KERNEL_FUNCTION(scene_intersect)
DEFINE_SPLIT_KERNEL_FUNCTION(lamp_emission)
DEFINE_SPLIT_KERNEL_FUNCTION(do_volume)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(queue_enqueue, QueueEnqueueLocals)
DEFINE_SPLIT_KERNEL_FUNCTION(indirect_background)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(shader_setup, uint)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(shader_sort, ShaderSortLocals)
DEFINE_SPLIT_KERNEL_FUNCTION(shader_eval)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(holdout_emission_blurring_pathtermination_ao, BackgroundAOLocals)
DEFINE_SPLIT_KERNEL_FUNCTION(subsurface_scatter)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(direct_lighting, uint)
DEFINE_SPLIT_KERNEL_FUNCTION(shadow_blocked_ao)
DEFINE_SPLIT_KERNEL_FUNCTION(shadow_blocked_dl)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(enqueue_inactive, uint)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(next_iteration_setup, uint)
DEFINE_SPLIT_KERNEL_FUNCTION(indirect_subsurface)
DEFINE_SPLIT_KERNEL_FUNCTION_LOCALS(buffer_update, uint)

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

#endif

