#include "hip/hip_runtime.h"
/* SPDX-License-Identifier: Apache-2.0
 * Copyright 2011-2022 Blender Foundation */

#define WITH_OSL

/* Copy of the regular OptiX kernels with additional OSL support. */

#include "kernel/device/optix/kernel_shader_raytrace.cu"

#include "kernel/bake/bake.h"
#include "kernel/integrator/shade_background.h"
#include "kernel/integrator/shade_light.h"
#include "kernel/integrator/shade_shadow.h"
#include "kernel/integrator/shade_volume.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_background()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_background(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_light()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_light(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_surface(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_volume()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_volume(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_shadow()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_shadow(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_displace()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_displace_evaluate(nullptr, input, output, global_index);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_background()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_background_evaluate(nullptr, input, output, global_index);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_curve_shadow_transparency()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_curve_shadow_transparency_evaluate(nullptr, input, output, global_index);
}
