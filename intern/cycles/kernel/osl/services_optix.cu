/* SPDX-License-Identifier: Apache-2.0
 * Copyright 2011-2022 Blender Foundation */

#define WITH_OSL

// clang-format off
#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/device/gpu/image.h"  /* Texture lookup uses normal CUDA intrinsics. */

#include "kernel/osl/services_gpu.h"
// clang-format on

extern "C" __device__ void __direct_callable__dummy_services()
{
}
